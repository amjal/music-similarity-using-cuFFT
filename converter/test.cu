#include<hip/hip_runtime.h>
#include<hipfft/hipfft.h>
#include<hipfft/hipfftXt.h>
#include<stdio.h>
#include<string>
#include<math.h>
hipfftComplex* read_file(std::string file_path, size_t * size, bool shrink){
	size_t size2 = 1000000;
	//*size = get_data_size(file_path);
	//shrink the sample into a power of 2 so that transformations are done fast
	//if(shrink)
	//	*size = (size_t)pow(2, (size_t)log2(*size));
	FILE* file;
	file = fopen(file_path.c_str(), "r");
	if(file == NULL){
		printf("Error: Couldn't open file %s\n", file_path.c_str());
		exit(EXIT_FAILURE);
	}
	hipfftComplex* data_cufft = (hipfftComplex*)malloc(*size*sizeof(hipfftComplex));
	hipfftComplex* chunk_cufft = (hipfftComplex*)malloc(size2*sizeof(hipfftComplex));
	int offset = 1000;
	unsigned char* data = (unsigned char*)malloc((*size +offset)*sizeof(char));
	fread(data, 1,(( *size)+offset),file);
	for(int i =0; i < *size; i ++){
		data_cufft[i].x = (float) data[i];	
		//we're dealing with real numbers so set phase to 0
		data_cufft[i].y = 0;
		if(i<size2){
			chunk_cufft[i].x = (float)data[i+offset];
			chunk_cufft[i].y = 0;
		}
		//printf("%f %f\n", data_cufft[i].x, chunk_cufft[i].x);

	}
	fclose(file);
		//getchar();

	hipfftHandle plan1;
	hipfftHandle plan2;
	hipfftPlan1d(&plan1, (int)*size, HIPFFT_R2C, 1);
	hipfftPlan1d(&plan2,(int)size2, HIPFFT_R2C, 1);

	hipfftComplex* d_data_cufft;
	hipfftComplex* d_chunk_cufft;
	hipMalloc((void**)&d_data_cufft, *size*sizeof(hipfftComplex));
	hipMalloc((void**)&d_chunk_cufft, size2*sizeof(hipfftComplex));
	hipMemcpy(d_data_cufft, data_cufft, *size*sizeof(hipfftComplex), hipMemcpyHostToDevice);
	hipMemcpy(d_chunk_cufft, chunk_cufft, size2*sizeof(hipfftComplex), hipMemcpyHostToDevice);
	hipfftExecR2C(plan1,(hipfftReal*)d_data_cufft, d_data_cufft);
	hipfftExecR2C(plan2,(hipfftReal*)d_chunk_cufft, d_chunk_cufft);
	hipMemcpy(data_cufft, d_data_cufft, (*size/2+1)*sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	hipMemcpy(chunk_cufft, d_chunk_cufft, (size2/2+1)*sizeof(hipfftComplex), hipMemcpyDeviceToHost);
	for(int i =0; i <(size2/2+1); i ++){
		printf("x1:%f , y1:%f, abs: %f\n", data_cufft[i].x, data_cufft[i].y, sqrt(data_cufft[i].x*data_cufft[i].x + data_cufft[i].y*data_cufft[i].y));
		printf("x2:%f , y2:%f, abs: %f\n", chunk_cufft[i].x, chunk_cufft[i].y, sqrt(chunk_cufft[i].x*chunk_cufft[i].x + chunk_cufft[i].y*chunk_cufft[i].y));
		printf("\n");
	}
	return data_cufft;
}
int main(int argc, char* argv[]){
	size_t s = 1000000;
	read_file(argv[1], &s, false);
	return 0;
}
