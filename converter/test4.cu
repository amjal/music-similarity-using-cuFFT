#include<hip/hip_runtime.h>
#include<stdio.h>
__global__
void something(int* a){
	int id = blockDim.x*blockIdx.x + threadIdx.x;
	a[id] = 0;
}
int main(){
	int * a;
	hipStream_t stream;
	hipStreamCreate(&stream);
	hipMalloc((void**)&a, 2049*sizeof(int));
	something<<<3,1024, 0, stream>>>(a);
	hipFree(a);
	return 0;
}
